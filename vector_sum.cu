
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define N 10

__global__ void add(const int *a, const int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Use std::vector for host arrays
    std::vector<int> a(N), b(N), c(N);

    int *dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;

    // Allocate memory on the GPU
    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));

    // Fill the arrays
    for (int i = 0; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy the arrays to the GPU
    hipMemcpy(dev_a, a.data(), N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b.data(), N * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // Copy the result back to the CPU
    hipMemcpy(c.data(), dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display the results using C++ streams
    for (int i = 0; i < N; ++i) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

    // Free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
