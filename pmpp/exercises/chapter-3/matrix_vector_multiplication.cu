#include <iostream>
#include <hip/hip_runtime.h>

// matrix multiplication kernel
__global__ void Matrix_Vector_Multiplication(float* A, const float* B, const float* C, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N) {
        float sum = 0.0f;
        for (int j = 0; j < N; j++) {
            sum += B[row * N + j] * C[j];
        }
        A[row] = sum;
    }
}

int main() {
    int N = 1024;
    size_t bytes = N * N * sizeof(float);
    size_t bytes_vector = N * sizeof(float);

    // allocate memory
    float *h_A = (float*)malloc(bytes_vector);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes_vector);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // initialize matrix
    for (int i = 0; i < N * N; ++i) {
        h_B[i] = static_cast<float>(i);
    }

    // initialize vector
    for (int i = 0; i < N; ++i) {
        h_C[i] = static_cast<float>(i);
    }

    // copy from host to device
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice);

    // launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    Matrix_Vector_Multiplication<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // copy data from device to host
    hipMemcpy(h_A, d_A, bytes_vector, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << "\n";
    }

    // free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
}
