#include <iostream>
#include <hip/hip_runtime.h>

// produces one output matrix column
__global__ void MatrixAdditionColumn(const float* B, const float* C, float* A, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < N) {
        for (int row = 0; row < N; ++row) {
            int idx = row * N + col; // column-major order
            A[idx] = B[idx] + C[idx];
        }
    }
}


int main() {
    int N = 1024;
    size_t bytes = N * N * sizeof(float);

    // allocate memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // initialize matrices
    for (int i = 0; i < N * N; ++i) {
        h_B[i] = static_cast<float>(i);
        h_C[i] = static_cast<float>(i);
    }

    // copy data from host to device
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, bytes, hipMemcpyHostToDevice);

    // Launch kernel for column-wise addition
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    MatrixAdditionColumn<<<blocksPerGrid, threadsPerBlock>>>(d_B, d_C, d_A, N);

    // copy data from device to host
    hipMemcpy(h_A, d_A, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << "\n";
    }

    // free up memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

}
